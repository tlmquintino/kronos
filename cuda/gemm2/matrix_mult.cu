#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

// #include <hip/hip_runtime.h>
// #include <hip/hip_runtime.h>

#include "matrix_sizes.h"
#include "matrix_mult.h"

#if 0 // version 1
// CUDA Kernel
__global__ void
cudakernel_matrix_mul( real_t* C, real_t* A, real_t* B, int wA, int wB)
{

   // 2D Thread ID
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   // value stores the element that is
   // computed by the thread
   real_t value = 0;
   for (int i = 0; i < wA; ++i)
   {
      real_t elementA = A[ty * wA + i];
      real_t elementB = B[i * wB + tx];
      value += elementA * elementB;
   }

   // Write the matrix to device memory each
   // thread writes one element
   C[ty * wA + tx] = value;
}
#endif

#if 1 // version 2
// CUDA Kernel
// Multiply two matrices A * B = C
__global__ void
cudakernel_matrix_mul( real_t* C, real_t* A, real_t* B, int wA, int wB)
{

   // 2D Thread ID
   int tx = blockIdx.x * TILE_SIZE + threadIdx.x;
   int ty = blockIdx.y * TILE_SIZE + threadIdx.y;

   // value stores the element that is
   // computed by the thread
   real_t value = 0;
   for (int i = 0; i < wA; ++i)
   {
     real_t elementA = A[ty * wA + i];
     real_t elementB = B[i * wB + tx];
      value += elementA * elementB;
   }

   // Write the matrix to device memory
   // each thread writes one element
   C[ty * wA + tx] = value;
}
#endif

void gpu_mat_mul(real_t* h_A, real_t* h_B, real_t* h_C )
{

    // allocate device memory
    real_t* d_A;
    real_t* d_B;
    real_t* d_C;

    unsigned int size_A = WA * HA;
    unsigned int size_B = WB * HB;
    unsigned int size_C = WC * HC;

    unsigned int mem_size_A = sizeof(real_t) * size_A;
    unsigned int mem_size_B = sizeof(real_t) * size_B;
    unsigned int mem_size_C = sizeof(real_t) * size_C;

    hipMalloc((void**) &d_A, mem_size_A);
    hipMalloc((void**) &d_B, mem_size_B);
    hipMalloc((void**) &d_C, mem_size_C);

    // copy host memory to device*/
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);
 
    // perform the calculation

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);
 
    //   execute the kernel
    cudakernel_matrix_mul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
 
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
 
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
